#include <pybind11/pybind11.h>
#include "saxpy_header_only.hpp"

PYBIND11_MODULE(cpp_bindings_header_only, m) {
    m.doc() = "pybind11 C++ bindings"; // optional module docstring

    m.def("saxpy_template", [](float a, long x_begin, long x_end, long y_begin, long output_begin){
            saxpy_template(a, reinterpret_cast<float const*>(x_begin), reinterpret_cast<float const*>(x_end), reinterpret_cast<float const*>(y_begin), reinterpret_cast<float *>(output_begin));
            }, "Compute a * x + y");
}
